//#define DEBUG

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#ifdef DEBUG
cudaError_t status;
void checkCuda(cudaError_t& status) {
    status = cudaGetLastError();
    if (status == cudaSuccess) {
        fprintf(stderr, "Success!\n");
    } else {
        fprintf(stderr, "CUDA error: %s\n", cudaGetErrorString(status));
        exit(-1);
    }
}
#endif

__global__ void CUDACross(bool *candidates, int size){
    for (int idx = blockIdx.x*blockDim.x + threadIdx.x; idx < size/2 + 1; idx += blockDim.x * gridDim.x) {
        int multiplier = idx + 2;
        int check = multiplier * multiplier; // bang when `multiplier` reaches ceil(sqrt(2^31)) = 46341
        //if (candidates[multiplier-2]) {    // which is when `N` gets to (46341-2-1)*2 + 2 = 92678
            while (check < size + 2){
                candidates[check - 2] = false;
                check += multiplier;
            }
        //}
    }
}

void init(bool *candidates, int size){
    for (int i = 0; i<size; i++)
        candidates[i] = true;
}

int main(int argc, char* argv[]) {
    /*if (argc != 2 || atoi(argv[1]) < 2 || atoi(argv[1]) > 1000000) {
        fprintf(stderr, "bad input\nusage: $ ./seqgenprimes N\nwhere N is in [2, 1000000]");
        exit(-1);
    }*/
    int N = atoi(argv[1]);
    int size = N - 1;

    bool* candidates = new bool[size];

    init(candidates, size);

    int deviceNum = 0;
    hipSetDevice(deviceNum);
    struct hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, deviceNum);

    int dimBlock = prop.maxThreadsPerBlock / 4;
    int dimGrid = prop.multiProcessorCount * 32;

#ifdef DEBUG
    fprintf(stderr, "maxThreadsPerBlock is %d\n", prop.maxThreadsPerBlock);
    fprintf(stderr, "maxThreadsPerMultiProcessor is %d\n", prop.maxThreadsPerMultiProcessor);
    fprintf(stderr, "totalGlobalMem is %d\n", prop.totalGlobalMem);
#endif

    //Initialize arrays
    bool *gpudata;

    //Allocate memory
    hipMalloc((void**)&gpudata, sizeof(bool)*size);
#ifdef DEBUG
    fprintf(stderr, "checking cudaMalloc()...\n");
    checkCuda(status);
#endif

    //Copy to GPU
    hipMemcpy(gpudata, candidates, sizeof(bool)*size, hipMemcpyHostToDevice);
#ifdef DEBUG
    fprintf(stderr, "checking cudaMemcpy() host to device...\n");
    checkCuda(status);
#endif

    //Kernel call on the GPU
//    CUDACross<<<bNum, tNum>>>(gpudata, size, bNum, tNum);
    CUDACross<<<dimGrid, dimBlock>>>(gpudata, size);
//    CUDACross<<<dimGrid, dimBlock>>>(gpudata, size, N);
#ifdef DEBUG
    fprintf(stderr, "checking kernel...\n");
    checkCuda(status);
#endif

    //Copy from GPU back onto host
    hipMemcpy(candidates, gpudata, sizeof(bool)*size, hipMemcpyDeviceToHost);
#ifdef DEBUG
    fprintf(stderr, "checking cudaMemcpy() device to host...\n");
    checkCuda(status);
#endif

    //Free the memory on the GPU
    hipFree(gpudata);

    char filename[20];
    sprintf(filename, "%d.txt", N);
    FILE *fp = fopen(filename, "w");
    fprintf(fp, "%d ", 2);
#ifdef DEBUG
    fprintf(stderr, "%d ", 2);
#endif
    for (int i = 1; i < size; ++i) {
        if (candidates[i]) fprintf(fp, "%d ", i+2);
#ifdef DEBUG
        if (candidates[i]) fprintf(stderr, "%d ", i+2);
#endif
    }
    return 0;
}
